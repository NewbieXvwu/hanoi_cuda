#include "hip/hip_runtime.h"
#include <windows.h>
#include <stdio.h>
#include <algorithm>
#include <string.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d (%s)\n", \
        __FILE__, __LINE__, err, hipGetErrorString(err)); \
        exit(1); \
    } \
}

// 使用常量内存存储频繁访问的只读参数
__constant__ int const_n;
__constant__ int const_direction;

__global__ void hanoi_kernel(int *d_steps, long long base, long long chunk_steps) {
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= chunk_steps) return;

    long long m = base + idx + 1;
    int disk = __ffsll(m) - 1;
    int from = (disk % 2) ? 1 : 0;
    
    // 使用预计算的const_direction替代实时计算
    int to = (from + const_direction * ((m >> (disk + 1)) % 2 ? 1 : 2)) % 3;
    from %= 3;
    to %= 3;
    
    // 使用预计算的const_n替代实时计算
    from = (from + (const_n % 2)) % 3;
    to = (to + (const_n % 2)) % 3;
    
    d_steps[idx] = (from << 4) | to;
}

// 进度条生成辅助函数
const char* generate_progress_bar(long long current, long long total) {
    static char bar[21];
    memset(bar, ' ', 20);
    bar[20] = '\0';
    
    int progress = (int)(20.0 * current / total);
    for (int i = 0; i < progress && i < 20; ++i) {
        bar[i] = '=';
    }
    if (progress < 20) {
        bar[progress] = '>';
    }
    return bar;
}

double get_time() {
    LARGE_INTEGER freq, time;
    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&time);
    return (double)time.QuadPart / freq.QuadPart;
}

void solve_hanoi(int n) {
    long long total_steps = (1LL << n) - 1;
    const double start_time = get_time(); // 记录总开始时间
    
    // 提前声明所有可能被跳过的变量
    float milliseconds = 0;  // 初始化为默认值
    double elapsed_sec = 0;
    double est_total = 0;
    double remaining_sec = 0;
    
    if (n > 40) {
        fprintf(stderr, "Error: Maximum supported layers is 40\n");
        exit(1);
    }

    // 预计算常量并拷贝到常量内存
    int host_direction = (n % 2) ? -1 : 1;
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_n), &n, sizeof(int)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_direction), &host_direction, sizeof(int)));

    // 复用显存指针（保留static优化）
    static int *d_steps = nullptr;
    static size_t allocated_size = 0;

    // 使用异步流进行内存操作
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    CHECK_CUDA(hipEventRecord(start));
    
    // 初始化颜色支持（Windows需要启用VT支持）
    #ifdef _WIN32
    HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
    DWORD mode = 0;
    GetConsoleMode(hConsole, &mode);
    SetConsoleMode(hConsole, mode | ENABLE_VIRTUAL_TERMINAL_PROCESSING);
    #endif

    for (long long base = 0; base < total_steps; ) {
        size_t free_mem, total_mem;
        
        // 减少显存查询频率（每10次循环查询一次）
        if (base % 10 == 0) {
            CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
        }
        
        // 使用更激进的内存分配策略（保留512MB安全边界）
        const size_t safety_margin = 512LL << 20;
        size_t available_mem = (free_mem > safety_margin) ? (free_mem - safety_margin) : 0;
        
        if (available_mem < sizeof(int)) {
            fprintf(stderr, "Insufficient GPU memory: Free %.2fGB < 512MB required\n", 
                   free_mem/1024.0/1024/1024);
            goto cleanup;
        }
        
        // 计算最优块大小（调整为256线程/块）
        const int threads = 256;
        long long max_chunk_steps = available_mem / sizeof(int);
        long long remaining_steps = total_steps - base;
        long long chunk_steps = std::min(max_chunk_steps, remaining_steps);
        
        // 动态调整块大小（最小1M，最大1G）
        chunk_steps = std::min(chunk_steps, 1LL << 30);  // 上限1G
        chunk_steps = std::max(chunk_steps, 1LL << 20);  // 下限1M

        // 自适应更新频率
        long long update_interval = std::max(total_steps / 1000, 1LL << 20);
        chunk_steps = std::min(chunk_steps, update_interval);

        // 复用显存（仅当需要更大内存时才重新分配）
        if (chunk_steps > allocated_size) {
            if (d_steps) CHECK_CUDA(hipFree(d_steps));
            CHECK_CUDA(hipMallocAsync(&d_steps, chunk_steps * sizeof(int), stream));
            allocated_size = chunk_steps;
        }
        
        // 使用异步内核启动
        dim3 blocks((chunk_steps + threads - 1) / threads);
        hanoi_kernel<<<blocks, threads, 0, stream>>>(d_steps, base, chunk_steps);
        
        // 进度显示逻辑
        elapsed_sec = (get_time() - start_time);
        est_total = (elapsed_sec * total_steps) / (base + 1);
        remaining_sec = est_total - elapsed_sec;
        
        printf("\r\x1b[36mProgress:\x1b[0m [\x1b[32m%-20s\x1b[0m] \x1b[33m%6.2f%%\x1b[0m | "
              "Chunk: \x1b[35m%5.2fGB\x1b[0m | "
              "Remaining: \x1b[31m%6.2fGB\x1b[0m | "
              "ETA: \x1b[34m%.1f sec\x1b[0m   ",
              generate_progress_bar(base, total_steps),
              base * 100.0 / total_steps,
              chunk_steps * sizeof(int) / (1024.0 * 1024 * 1024),
              (total_steps - base) * sizeof(int) / (1024.0 * 1024 * 1024),
              remaining_sec);
        fflush(stdout);

        CHECK_CUDA(hipStreamSynchronize(stream));
        base += chunk_steps;
    }
    
    // 完成时换行
    printf("\n");
    
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

cleanup:
    // 现在可以安全访问所有变量
    if (milliseconds > 0) {
        printf("\x1b[32mPure GPU compute time: %.2f ms\x1b[0m\n", milliseconds);
    }
    
    // 显存释放策略：小规模计算立即释放，大规模计算保留复用
    if (d_steps && n < 30) {  // 30层以下立即释放
        CHECK_CUDA(hipFree(d_steps));
        d_steps = nullptr;
        allocated_size = 0;
    }
    
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipStreamDestroy(stream));
}

int main() {
    int n;
    printf("Enter number of Hanoi layers: ");
    scanf("%d", &n);

    if (n < 1) {
        printf("Number of layers must be at least 1\n");
        return 1;
    }

    double start = get_time();
    solve_hanoi(n);
    double end = get_time();

    printf("Total time: %.2f seconds\n", end - start);

    printf("\nPress any key to exit...");
    getchar();
    getchar();

    return 0;
}